
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


__global__ void getMaxUsingAtomicFunction(int* in) {
    atomicMax(&in[0], in[blockIdx.x * blockDim.x + threadIdx.x]);
}


__global__ void getMaxUsingReduction(int* in, int size) {
    int step = 1;
    unsigned int i = (blockIdx.x * blockDim.x + threadIdx.x);
    while(i % step == 0 && step < size) {
//        printf("Comparing: %d and %d, step: %d on i: %d\n", 2*i, 2*i + step, step, i);
        atomicMax(&in[2*i], in[2*i + step]);
        step *= 2;
        __syncthreads();
    }
}

__global__ void getMaxUsingReductionFast(int* in, int size) {
    int step = 1;
    unsigned int i = (blockIdx.x * blockDim.x + threadIdx.x);
    while(i % step == 0 && step < size) {
//        printf("Comparing: %d and %d, step: %d on i: %d\n", 2*i, 2*i + step, step, i);
//        atomicMax(&in[2*i], in[2*i + step]);
        if(in[2*i] < in[2*i + step]) in[2*i] = in[2*i + step];
        step *= 2;
        __syncthreads();
    }
}

void findMaxUsingCPU(int size) {
    int* temp = (int*)(malloc(size * sizeof(int)));
    for(int i = 0; i < size; i++) {
        temp[i] = i;
    }

    const auto startc = std :: chrono :: steady_clock :: now () ;

    int biggest = -INFINITY;
    for(int i = 0; i < size; i++) {
        if(biggest < temp[i]){
            biggest = temp[i];
        }
    }

    const auto end = std :: chrono :: steady_clock :: now () ;
    const std :: chrono :: duration<double> elapsed_seconds{end - startc};
    std::cout << size << ";" << elapsed_seconds.count()*1000 << ";";

    free(temp);
}

void findMaxUsingReduction(int size) {
    int* temp = (int*)(malloc(size * sizeof(int)));
    for(int i = 0; i < size; i++) {
        temp[i] = i;
    }

    int* input;
    hipMalloc(&input, size*sizeof(int));
    hipMemcpy( input, temp, size*sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start);


    getMaxUsingReduction<<<1, size / 2>>>(input, size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << size << ";" << milliseconds << std::endl;
//    cudaMemcpy( temp, input, size * sizeof(int), cudaMemcpyDeviceToHost);
//    std::cout << "Results: " << std::endl;
//    for(int i = 0; i < size; i++) std::cout << temp[i] << ", ";
//    std::cout << std::endl;

    free(temp);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(input);
}

void findMaxUsingReductionFast(int size) {
    int* temp = (int*)(malloc(size * sizeof(int)));
    for(int i = 0; i < size; i++) {
        temp[i] = i;
    }

    int* input;
    hipMalloc(&input, size*sizeof(int));
    hipMemcpy( input, temp, size*sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start);


    getMaxUsingReductionFast<<<1, size / 2>>>(input, size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << size << ";" << milliseconds << std::endl;
//    cudaMemcpy( temp, input, size * sizeof(int), cudaMemcpyDeviceToHost);
//    std::cout << "Results: " << std::endl;
//    for(int i = 0; i < size; i++) std::cout << temp[i] << ", ";
//    std::cout << std::endl;

    free(temp);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(input);
}

void findMaxUsingAtomics(int size) {
    int* temp = (int*)(malloc(size * sizeof(int)));
    for(int i = 0; i < size; i++) {
        temp[i] = i;
    }

    int* input;
    hipMalloc(&input, size*sizeof(int));
    hipMemcpy( input, temp, size*sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start);
    const auto startc = std :: chrono :: steady_clock :: now () ;

    getMaxUsingAtomicFunction<<<1, size>>>(input);

    const auto end = std :: chrono :: steady_clock :: now () ;

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);
    const std :: chrono :: duration<double> elapsed_seconds{end - startc};

    std::cout << size << ";" << milliseconds << ";";
//    cudaMemcpy( temp, input, size * sizeof(int), cudaMemcpyDeviceToHost);
//    std::cout << "Results: " << std::endl;
//    for(int i = 0; i < size; i++) std::cout << temp[i] << ", ";
//    std::cout << std::endl;

    free(temp);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(input);
}

int main() {
//    int size = 2000;
    findMaxUsingCPU(1);
    findMaxUsingAtomics(1);
    findMaxUsingReduction(1);
    findMaxUsingReductionFast(1);

    for(int size = 1; size < 2048; size++) {
        findMaxUsingCPU(size);
        findMaxUsingAtomics(size);
        findMaxUsingReduction(size);
        findMaxUsingReductionFast(size);
    }
    return 0;
}



